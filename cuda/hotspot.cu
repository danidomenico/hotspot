#include "hip/hip_runtime.h"
%%writefile hotspot.cu

#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <cassert>
#include <time.h>
#include <sys/time.h>

using FLOAT = float;

// Returns the current system time in microseconds 
long long get_time() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return (tv.tv_sec * 1000000) + tv.tv_usec;
}

#ifdef RD_WG_SIZE_0_0                                                            
        #define BLOCK_SIZE RD_WG_SIZE_0_0                                        
#elif defined(RD_WG_SIZE_0)                                                      
        #define BLOCK_SIZE RD_WG_SIZE_0                                          
#elif defined(RD_WG_SIZE)                                                        
        #define BLOCK_SIZE RD_WG_SIZE                                            
#else                                                                                    
        #define BLOCK_SIZE 16                                                            
#endif                                                                                   

#define STR_SIZE 256

/* maximum power density possible (say 300W for a 10mm x 10mm chip)	*/
#define MAX_PD	(3.0e6)
/* required precision in degrees	*/
#define PRECISION	0.001
#define SPEC_HEAT_SI 1.75e6
#define K_SI 100
/* capacitance fitting factor	*/
#define FACTOR_CHIP	0.5

/* chip parameters	*/
float t_chip = 0.0005;
float chip_height = 0.016;
float chip_width = 0.016;
/* ambient temperature, assuming no package at all	*/
float amb_temp = 80.0;

/* define timer macros */
#define pin_stats_reset()   startCycle()
#define pin_stats_pause(cycles)   stopCycle(cycles)
#define pin_stats_dump(cycles)    printf("timer: %Lu\n", cycles)

#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )
#define MIN(a, b) ((a)<=(b) ? (a) : (b))

__global__ void calculate_temp(int iteration,  //number of iteration
                               float *power,   //power input
                               float *temp_src,    //temperature input/output
                               float *temp_dst,    //temperature input/output
                               int grid_cols,  //Col of grid
                               int grid_rows,  //Row of grid
							   int border_cols,  // border offset 
							   int border_rows,  // border offset
                               float Cap,      //Capacitance
                               float Rx, 
                               float Ry, 
                               float Rz, 
                               float step, 
                               float time_elapsed){
	
        __shared__ float temp_on_cuda[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float power_on_cuda[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float temp_t[BLOCK_SIZE][BLOCK_SIZE]; // saving temparary temperature result

	float amb_temp = 80.0;
        float step_div_Cap;
        float Rx_1,Ry_1,Rz_1;
        
	int bx = blockIdx.x;
        int by = blockIdx.y;

	int tx=threadIdx.x;
	int ty=threadIdx.y;
	
	step_div_Cap=step/Cap;
	
	Rx_1=1/Rx;
	Ry_1=1/Ry;
	Rz_1=1/Rz;
	
        // each block finally computes result for a small block
        // after N iterations. 
        // it is the non-overlapping small blocks that cover 
        // all the input data

        // calculate the small block size
	int small_block_rows = BLOCK_SIZE-iteration*2;//EXPAND_RATE
	int small_block_cols = BLOCK_SIZE-iteration*2;//EXPAND_RATE

        // calculate the boundary for the block according to 
        // the boundary of its small block
        int blkY = small_block_rows*by-border_rows;
        int blkX = small_block_cols*bx-border_cols;
        int blkYmax = blkY+BLOCK_SIZE-1;
        int blkXmax = blkX+BLOCK_SIZE-1;

        // calculate the global thread coordination
	int yidx = blkY+ty;
	int xidx = blkX+tx;

        // load data if it is within the valid input range
	int loadYidx=yidx, loadXidx=xidx;
        int index = grid_cols*loadYidx+loadXidx;
       
	if(IN_RANGE(loadYidx, 0, grid_rows-1) && IN_RANGE(loadXidx, 0, grid_cols-1)){
            temp_on_cuda[ty][tx] = temp_src[index];  // Load the temperature data from global memory to shared memory
            power_on_cuda[ty][tx] = power[index];// Load the power data from global memory to shared memory
	}
	__syncthreads();

        // effective range within this block that falls within 
        // the valid range of the input data
        // used to rule out computation outside the boundary.
        int validYmin = (blkY < 0) ? -blkY : 0;
        int validYmax = (blkYmax > grid_rows-1) ? BLOCK_SIZE-1-(blkYmax-grid_rows+1) : BLOCK_SIZE-1;
        int validXmin = (blkX < 0) ? -blkX : 0;
        int validXmax = (blkXmax > grid_cols-1) ? BLOCK_SIZE-1-(blkXmax-grid_cols+1) : BLOCK_SIZE-1;

        int N = ty-1;
        int S = ty+1;
        int W = tx-1;
        int E = tx+1;
        
        N = (N < validYmin) ? validYmin : N;
        S = (S > validYmax) ? validYmax : S;
        W = (W < validXmin) ? validXmin : W;
        E = (E > validXmax) ? validXmax : E;

        bool computed;
        for (int i=0; i<iteration ; i++){ 
            computed = false;
            if( IN_RANGE(tx, i+1, BLOCK_SIZE-i-2) &&  \
                  IN_RANGE(ty, i+1, BLOCK_SIZE-i-2) &&  \
                  IN_RANGE(tx, validXmin, validXmax) && \
                  IN_RANGE(ty, validYmin, validYmax) ) {
                  computed = true;
                  temp_t[ty][tx] =   temp_on_cuda[ty][tx] + step_div_Cap * (power_on_cuda[ty][tx] + 
	       	         (temp_on_cuda[S][tx] + temp_on_cuda[N][tx] - 2.0*temp_on_cuda[ty][tx]) * Ry_1 + 
		             (temp_on_cuda[ty][E] + temp_on_cuda[ty][W] - 2.0*temp_on_cuda[ty][tx]) * Rx_1 + 
		             (amb_temp - temp_on_cuda[ty][tx]) * Rz_1);
	
            }
            __syncthreads();
            if(i==iteration-1)
                break;
            if(computed)	 //Assign the computation range
                temp_on_cuda[ty][tx]= temp_t[ty][tx];
            __syncthreads();
          }

      // update the global memory
      // after the last iteration, only threads coordinated within the 
      // small block perform the calculation and switch on ``computed''
      if (computed){
          temp_dst[index]= temp_t[ty][tx];		
      }
}

/*
   compute N time steps
*/

int compute_tran_temp(FLOAT *MatrixPower, FLOAT *MatrixTemp[2], int col, int row, \
		int total_iterations, int num_iterations, int blockCols, int blockRows, int borderCols, int borderRows) {
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(blockCols, blockRows);  
	
	float grid_height = chip_height / row;
	float grid_width = chip_width / col;

	float Cap = FACTOR_CHIP * SPEC_HEAT_SI * t_chip * grid_width * grid_height;
	float Rx = grid_width / (2.0 * K_SI * t_chip * grid_height);
	float Ry = grid_height / (2.0 * K_SI * t_chip * grid_width);
	float Rz = t_chip / (K_SI * grid_height * grid_width);

	float max_slope = MAX_PD / (FACTOR_CHIP * t_chip * SPEC_HEAT_SI);
	float step = PRECISION / max_slope;

    float time_elapsed = 0.001;

    int src = 1, dst = 0;
	
	for(int t = 0; t < total_iterations; t+=num_iterations) {
        int temp = src;
        src = dst;
        dst = temp;
        calculate_temp<<<dimGrid, dimBlock>>>(MIN(num_iterations, total_iterations-t), MatrixPower, MatrixTemp[src], MatrixTemp[dst],\
		    col,row, borderCols, borderRows, Cap, Rx, Ry, Rz, step, time_elapsed);
	}
    
    return dst;
}

void fatal(std::string s) {
	std::cerr << "Hotspot error: " << s << std::endl;
	std::abort();
}

void generate_input(FLOAT* temp, FLOAT* power, FLOAT* result, int order_size) {
	const FLOAT MIN = 323.0;
	const FLOAT MAX = 341.0;
	FLOAT value = MIN;
	bool  asc = true;
	
	for(unsigned i=0; i<order_size * order_size; i++) {
		//POWER
		FLOAT x = (rand() % 1000000) / 1000000.0;
		power[i] = x;
		
		//TEMP
		x = (rand() % 1000000) / 1000000.0;
		if(asc) {
			if((value + x) > MAX) {
				value -= x;
				asc   = false;
			} else
				value += x;
		} else {
			if((value - x) < MIN) {
				value += x;
				asc   = true;
			} else
				value -= x;
		}
		temp[i] = value;
		
		//RESULT
		result[i] = 0.0;
	}
}

void usage(int argc, char **argv) {
	std::cerr << "Usage:" << argv[0] << "<order> <iterations>\n";
	std::cerr << "\t<order>      - order for the grid - size= <order>X<order> (positive integer)\n";
	std::cerr << "\t<iterations> - number of iterations\n";
    std::cerr << "\t<pyramid_height> - pyramid heigh\n";
	std::abort();
}

int main(int argc, char** argv) {
    printf("WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);

    int order_size, iterations;
    int pyramid_height = 1; // number of iterations
	FLOAT *temp, *power, *result;
	
	/* check validity of inputs*/
	if(argc != 4)
		usage(argc, argv);
	if( (order_size = atoi(argv[1])) <= 0 ||
		(iterations = atoi(argv[2])) <= 0 ||
        (pyramid_height = atoi(argv[3])) <= 0)  
		usage(argc, argv);
    
    /* --------------- pyramid parameters --------------- */
    # define EXPAND_RATE 2// add one iteration will extend the pyramid base by 2 per each borderline
    int borderCols = (pyramid_height)*EXPAND_RATE/2;
    int borderRows = (pyramid_height)*EXPAND_RATE/2;
    int smallBlockCol = BLOCK_SIZE-(pyramid_height)*EXPAND_RATE;
    int smallBlockRow = BLOCK_SIZE-(pyramid_height)*EXPAND_RATE;
    int blockCols = order_size/smallBlockCol+((order_size%smallBlockCol==0)?0:1);
    int blockRows = order_size/smallBlockRow+((order_size%smallBlockRow==0)?0:1);

    printf("pyramidHeight: %d\ngridSize: [%d, %d]\nborder:[%d, %d]\nblockGrid:[%d, %d]\ntargetBlock:[%d, %d]\n",\
	pyramid_height, order_size, order_size, borderCols, borderRows, blockCols, blockRows, smallBlockCol, smallBlockRow);

    /* allocate memory for the temperature and power arrays	*/
	temp   = new FLOAT[order_size * order_size];
	power  = new FLOAT[order_size * order_size];
	result = new FLOAT[order_size * order_size];
	if(!temp || !power || ! result)
		fatal("unable to allocate memory");
	
	/* generate input */
	generate_input(temp, power, result, order_size);

    float *MatrixTemp[2], *MatrixPower;
    hipMalloc((void**)&MatrixTemp[0], sizeof(FLOAT) * order_size * order_size);
    hipMalloc((void**)&MatrixTemp[1], sizeof(FLOAT) * order_size * order_size);
    hipMalloc((void**)&MatrixPower, sizeof(FLOAT) * order_size * order_size);

    long long start_time = get_time();
    hipMemcpy(MatrixTemp[0], temp, sizeof(FLOAT) * order_size * order_size, hipMemcpyHostToDevice);
    hipMemcpy(MatrixPower, power, sizeof(FLOAT) * order_size * order_size, hipMemcpyHostToDevice);
    
#ifdef VERBOSE
	printf("Start computing the transient temperature\n");
#endif
    
    int ret = compute_tran_temp(MatrixPower, MatrixTemp, order_size, order_size, \
	 iterations, pyramid_height, blockCols, blockRows, borderCols, borderRows);
	printf("Ending simulation\n");
    hipMemcpy(result, MatrixTemp[ret], sizeof(FLOAT)* order_size * order_size, hipMemcpyDeviceToHost);

    long long end_time = get_time();
  
    float total_time = ((float) (end_time - start_time)) / (1000*1000);

    //printf("BENCH=hotspot;backend=%s;size=%d;block_size=%d;iterations=%d;threads=%d;gpus=%d;time=%.8f\n", 
	printf("hotspot;%s;%d;%d;%d;%d;%d;%.8f\n",
		"CUDA", order_size, order_size, iterations, 0, 1, total_time);

    hipFree(MatrixPower);
    hipFree(MatrixTemp[0]);
    hipFree(MatrixTemp[1]);
    
    delete[] temp;
	delete[] power;
	delete[] result;
	
	return 0;
}
